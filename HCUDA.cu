#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <hipblas.h>
#include <math.h>
#include <assert.h>
#include "fp16_conversion.h"

extern "C" {
#include "HMem.h"
}

#define CEIL(x,y) (((x)+(y)-1) / (y))

#define CHECKCUDASTATUS(status, invoker) {                                                    \
    if (status != hipSuccess) printf("ERROR %s: %s", invoker, hipGetErrorString(status)); \
}

#define CHECKCUBLASSTATUS(status, invoker, message) {                                \
    if (status != HIPBLAS_STATUS_SUCCESS) printf("ERROR %s: %s", invoker, message); \
}

#define CHECKCUDNNSTATUS(status, invoker) {                                                        \
    if (status != HIPDNN_STATUS_SUCCESS) printf("ERROR %s: %s", invoker, hipdnnGetErrorString(status)); \
}

#define CHECKCURANDSTATUS(status, invoker, message) {                                \
    if (status != HIPRAND_STATUS_SUCCESS) printf("ERROR %s: %s", invoker, message); \
}

#define THREADPERBLOCK 256  

static int GPUDevId = -1; 
static Boolean GPUInit = FALSE;
hipblasHandle_t cublasHandle;
#ifdef CUDNN
hipdnnHandle_t cudnnHandle;
hipdnnDataType_t dataTypeCUDNN;
hipdnnTensorFormat_t tensorFormatCUDNN;
hipdnnTensorDescriptor_t srcTensorDescCUDNN;
hipdnnTensorDescriptor_t dstTensorDescCUDNN;
hipdnnTensorDescriptor_t biasTensorDescCUDNN;
hipdnnActivationDescriptor_t actfunDescCUDNN;
#endif


#define GPU_DEV_ID 0
#define FP16MM

extern "C" {
    static void ShowAllGPUs(void) {
        int nGPU, i;
        hipDeviceProp_t prop;

        /*  */
        CHECKCUDASTATUS(hipGetDeviceCount(&nGPU), "ShowAllGPUs")
        if (nGPU == 0) {
            printf("ERROR ShowAllGPUs: No GPU device");
            exit(1);
        }
        /*  */
        for (i = 0; i < nGPU; ++i) {
            CHECKCUDASTATUS(hipGetDeviceProperties(&prop, i), "ShowAllGPUs")
            printf("GPU %d: %s, %luMB, SM = %d.%d", i, prop.name, prop.totalGlobalMem / 1048576, prop.major, prop.minor);
            if (GPUDevId == i)
                printf(" [Selected]");
            printf("\n");
        }
    }
}

extern "C" {
    void StartCUDA(void) {
        hipDeviceProp_t prop;

        /* initialize the library and device */
        if (!GPUInit) {
            /* select GPU device 0 */
            GPUDevId = GPU_DEV_ID;
            CHECKCUDASTATUS(hipSetDevice(GPUDevId), "InitCUDA")
            CHECKCUDASTATUS(hipGetDeviceProperties(&prop, GPUDevId), "InitCUDA")
            /* initiate CUBLAS */
            CHECKCUBLASSTATUS(hipblasCreate(&cublasHandle), "InitCUDA", "Fail to initialise CUBLAS")
            /* set GPUInit flag */
            GPUInit = TRUE;
            /* show devices */
            ShowAllGPUs();
    #ifdef CUDNN
    #ifdef DOUBLEANN
            dataTypeCUDNN = HIPDNN_DATA_DOUBLE;
    #else
            dataTypeCUDNN = HIPDNN_DATA_FLOAT;
    #endif
            tensorFormatCUDNN = HIPDNN_TENSOR_NCHW;
            CHECKCUDNNSTATUS(hipdnnCreate(&cudnnHandle), "StartCUDA")
            CHECKCUDNNSTATUS(hipdnnCreateTensorDescriptor(&srcTensorDescCUDNN), "StartCUDA")
            CHECKCUDNNSTATUS(hipdnnCreateTensorDescriptor(&dstTensorDescCUDNN), "StartCUDA")
            CHECKCUDNNSTATUS(hipdnnCreateTensorDescriptor(&biasTensorDescCUDNN), "StartCUDA")
            CHECKCUDNNSTATUS(hipdnnCreateActivationDescriptor(&actfunDescCUDNN), "StartCUDA")
    #endif

        }
        else
            printf("InitCUDA: GPU device %d already initialised", GPUDevId);

        printf("\n");
    }
}

extern "C" {
    void StopCUDA(void) {

        if (GPUInit) {
    #ifdef CUDNN
            CHECKCUDNNSTATUS(hipdnnDestroyTensorDescriptor(srcTensorDescCUDNN), "StopCUDA")
            CHECKCUDNNSTATUS(hipdnnDestroyTensorDescriptor(dstTensorDescCUDNN), "StopCUDA")
            CHECKCUDNNSTATUS(hipdnnDestroyTensorDescriptor(biasTensorDescCUDNN), "StopCUDA")
            CHECKCUDNNSTATUS(hipdnnDestroyActivationDescriptor(actfunDescCUDNN), "StopCUDA")
            CHECKCUDNNSTATUS(hipdnnDestroy(cudnnHandle), "StopCUDA")
    #endif
            /* destroy the context on the GPU */
            CHECKCUBLASSTATUS(hipblasDestroy(cublasHandle), "StopCUDA", "Fail to destroy CUBLAS")
            /* shutdown CUBLAS */
            CHECKCUDASTATUS(hipDeviceReset(), "StopCUDA");
            /* reset GPU IDs and the flag */
            GPUDevId = -1;
            GPUInit = FALSE;
        }
        else
            printf("StopCUDA: GPU device has already stopped");
    }
}


extern "C" {
    void SyncDev2Host(void *devPtr, void *hostPtr, size_t size) {
        hipMemcpy(hostPtr, devPtr, size, hipMemcpyDeviceToHost);
    }
}

extern "C" {
    void SyncHost2Dev(void *hostPtr, void *devPtr, size_t size) {
        hipMemcpy(devPtr, hostPtr, size, hipMemcpyHostToDevice);
    }
}

extern "C" {
    void DevDispose(void *devPtr, size_t size) {
        hipFree(devPtr);
    }
}

extern "C" {
    Boolean DevNew(void **devAddr, size_t size) {
        if (hipMalloc(devAddr, size) != hipSuccess)
            return FALSE;
        return TRUE;
    }
}

const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}


inline
hipblasStatus_t checkCublas(hipblasStatus_t result)
{
  if (result != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cublasGetErrorString(result));
    assert(result == HIPBLAS_STATUS_SUCCESS);
  }
  return result;
}

/*----------Kernel Functions----------*/

__global__
void HKern_SelfAddNSegment(NFloat *rhPtr, int segLen, NFloat *lhPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        lhPtr[pos] = lhPtr[pos] + rhPtr[pos];
        
        /*if (pos == 0) {
            printf("Size of half: %lu, Size of half2: %lu", sizeof(half), sizeof(half2));
        }*/
    }
}

__global__
void HKern_SelfAddNSegmentHalf(NFloat *rhPtr, int segLen, NFloat *lhPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        lhPtr[pos] = __half2float(__hadd(__float2half(lhPtr[pos]), __float2half(rhPtr[pos])));
    }
}

__global__
void HKern_MulMatrices(NFloat *lhPtr, NFloat *rhPtr, NFloat *resPtr, int N) {
    /* Assuming both matrices are N x N */

    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    float tmpSum = 0;

    if (col < N && row < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += lhPtr[row * N + i] * rhPtr[i * N + col];
        }
	resPtr[row * N + col] = tmpSum; 
    }
}

__global__
void HKern_MulMatrices_Half(NFloat *lhPtr, NFloat *rhPtr, NFloat *resPtr, int N) {
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //float tmpSum = 0;
    __half tmpSum = __int2half_rn(0);

    if (col < N && row < N) {
        for (int i = 0; i < N; i++) {
            //tmpSum += lhPtr[row * N + i] * rhPtr[i * N + col];
            tmpSum = __hadd(tmpSum, __hmul(__float2half(lhPtr[row * N + i]), __float2half(rhPtr[i * N + col])));
        }
	    resPtr[row * N + col] = __half2float(tmpSum);
    }
}



__global__
void HKern_MulMatrices_Half2(NFloat *lhPtr, NFloat *rhPtr, NFloat *resPtr, int N, hipblasHandle_t cublasHandle) {
    /*
    hipblasStatus_t stat;
    
    stat = hipblasHgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, , lda, d_B, ldb, beta, d_C, ldc);


    hipblasStatus_t hipblasHgemm(hipblasHandle_t handle,
                           hipblasOperation_t transa, hipblasOperation_t transb,
                           int m, int n, int k,
                           const __half *alpha,
                           const __half *A, int lda,
                           const __half *B, int ldb,
                           const __half *beta,
                           __half *C, int ldc)
    */
        
}


/*
__global__
void HKern_SelfMulNSegment(NFloat *rhPtr, int segLen, NFloat *lhPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
	lhPtr[pos] = lhPtr[pos] * rhPtr[pos];
    }
}
*/

/*
__global__
void HKern_MulNSegment(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) 
        resPtr[pos] = lhPtr[pos] * rhPtr[pos];
}
*/

/*----------C Wrappers----------*/

extern "C" {
    void SelfAddNSegmentCUDA(NFloat *rhPtr, int segLen, NFloat *lhPtr) {
        int nBlocks;

        nBlocks = CEIL(segLen, THREADPERBLOCK);
        /*if (nBlocks > MAXBLOCKNUM)
            HError(8890, (char *)"MulNSegmentCUDA: Block number exceeds the maximum");*/
        HKern_SelfAddNSegment<<<nBlocks, THREADPERBLOCK>>>(rhPtr, segLen, lhPtr);
    }
}

extern "C" {
    void SelfAddNSegmentCUDAHalf(NFloat *rhPtr, int segLen, NFloat *lhPtr) {
        int nBlocks;

        nBlocks = CEIL(segLen, THREADPERBLOCK);
        //if (nBlocks > MAXBLOCKNUM)
        //  HError(8890, (char *)"MulNSegmentCUDA: Block number exceeds the maximum");
        HKern_SelfAddNSegmentHalf<<<nBlocks, THREADPERBLOCK>>>(rhPtr, segLen, lhPtr);
    }
}

extern "C" {
    void MulMatricesCUDA(NFloat *lhPtr, NFloat *rhPtr, NFloat *resPtr, int N, int segLen) {
	    int nBlocks;

	    dim3 threadsPerBlock(16, 16);  // hard coded as 16 x 16 for now
	    nBlocks = CEIL(segLen, THREADPERBLOCK);
        /*if (nBlocks > MAXBLOCKNUM)
            HError(8890, (char *)"MulNSegmentCUDA: Block number exceeds the maximum");*/
	    dim3 blocksPerGrid(nBlocks, nBlocks);
	    HKern_MulMatrices<<<blocksPerGrid, threadsPerBlock>>>(lhPtr, rhPtr, resPtr, N);
    }
}

extern "C" {
     void MulMatricesCUDAHalf(NFloat *lhPtr, NFloat *rhPtr, NFloat *resPtr, int N, int segLen) {
	    int nBlocks;
	    dim3 threadsPerBlock(16, 16);  // NOTE: hard coded as 16 x 16 for now
	    nBlocks = CEIL(segLen, THREADPERBLOCK);
        //if (nBlocks > MAXBLOCKNUM)
        //    HError(8890, (char *)"MulNSegmentCUDA: Block number exceeds the maximum");
	    dim3 blocksPerGrid(nBlocks, nBlocks);

        //printf("nBlocks: %d, segLen: %d", nBlocks, segLen);
	    HKern_MulMatrices_Half<<<blocksPerGrid, threadsPerBlock>>>(lhPtr, rhPtr, resPtr, N);
    }
}

extern "C" {
     void MulMatricesCUDAHalf2(NFloat *lhPtr, NFloat *rhPtr, NFloat *resPtr, int N, int segLen) {
        /*
	    int nBlocks;
	    dim3 threadsPerBlock(16, 16);  // NOTE: hard coded as 16 x 16 for now
	    nBlocks = CEIL(segLen, THREADPERBLOCK);
        //if (nBlocks > MAXBLOCKNUM)
        //    HError(8890, (char *)"MulNSegmentCUDA: Block number exceeds the maximum");
	    dim3 blocksPerGrid(nBlocks, nBlocks);
        
        //printf("nBlocks: %d, segLen: %d", nBlocks, segLen);
	    HKern_MulMatrices_Half2<<<blocksPerGrid, threadsPerBlock>>>(lhPtr, rhPtr, resPtr, N);
        */

		hipblasStatus_t stat;
        hipblasHandle_t handle;
        
        checkCublas(hipblasCreate(&handle));

        __half *d_A, *d_B, *d_C;
        checkCuda(hipMallocManaged(&d_A, N * N * sizeof(__half)));
        checkCuda(hipMallocManaged(&d_B, N * N * sizeof(__half)));
        checkCuda(hipMallocManaged(&d_C, N * N * sizeof(__half)));
        
        for (int i = 0; i < N * N; i++) {
              d_A[i] = approx_float_to_half(lhPtr[i]);
          	  d_B[i] = approx_float_to_half(rhPtr[i]);
          	  d_C[i] = approx_float_to_half(resPtr[i]);
        }

        int lda, ldb, ldc, m, n, k;
        const __half alf = approx_float_to_half(1.0);
        const __half bet = approx_float_to_half(0.0);
        const __half *alpha = &alf;
        const __half *beta = &bet;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        double sum = 0.0;

        hipEventRecord(start, 0);
        m=n=k=N;
        lda = m;
        ldb = k;
        ldc = m;
        
        stat = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc);
    
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);

        if(stat != HIPBLAS_STATUS_SUCCESS){
            printf("hipblasHgemm failed");
            exit(1);
        }
        assert(!hipGetLastError());
      
        float elapsed;
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0f;
        sum += elapsed;

        printf("Time elapsed: %e", sum);

        for (int i = 0; i < N * N; i++) {
              d_A[i] = approx_float_to_half(lhPtr[i]);
          	  d_B[i] = approx_float_to_half(rhPtr[i]);
          	  d_C[i] = approx_float_to_half(resPtr[i]);
        }
        
        
        //Free GPU memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

    }
}

/*
extern "C" {
    void SelfMulNSegmentCUDA(NFloat *rhPtr, int segLen, NFloat *lhPtr) {
        int nBlocks;

        nBlocks = CEIL(segLen, THREADPERBLOCK);
        //if (nBlocks > MAXBLOCKNUM)
        //  HError(8890, (char *)"MulNSegmentCUDA: Block number exceeds the maximum");
        HKern_SelfMulNSegment<<<nBlocks, THREADPERBLOCK>>>(rhPtr, segLen, lhPtr);
    }
}

extern "C" {
	void MulNSegmentCUDA(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
        int nBlocks;

        nBlocks = CEIL(segLen, THREADPERBLOCK);
        //if (nBlocks > MAXBLOCKNUM)
        //  HError(8890, (char *)"MulNSegmentCUDA: Block number exceeds the maximum");
        HKern_MulNSegment<<<nBlocks, THREADPERBLOCK>>>(lhPtr, rhPtr, segLen, resPtr);
    }
}
*/
/*---------------------------END OF HCUDA_ext.cu---------------------------*/
